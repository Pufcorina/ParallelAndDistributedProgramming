
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

int main();

int* generatePolynom(const int size);

hipError_t multiplyWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void multiplyKernel(int *c, const int *a, const int *b, const int size)
{
    int i = threadIdx.x;
	for ( int j = 0; j < size; j++)
		c[i + j] += a[i] * b[j];
}

int main()
{
    const int arraySize = 5;
	const int* a = generatePolynom(arraySize);
	const int* b = generatePolynom(arraySize);
    int c[2 * arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = multiplyWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	for (int i = 0; i < arraySize; i++)
		printf("%d ", a[i]);

	printf("\n");

	for (int i = 0; i < arraySize; i++)
		printf("%d ", b[i]);

	printf("\n");

	for (int i = 0; i < 2 * arraySize - 1; i++)
		printf("%d ", c[i]);

	printf("\n");

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// in the range 1985-2014 
int * generatePolynom(const int size)
{
	int *pol = (int*) malloc(size * sizeof(int));
	for (int i = 0; i < size; i++)
		pol[i] = rand() % 30 + 1985;
	return pol;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t multiplyWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, 2 * size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    // Launch a kernel on the GPU with one thread for each element.
	multiplyKernel<<<1, size>>>(dev_c, dev_a, dev_b, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, 2 * size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
